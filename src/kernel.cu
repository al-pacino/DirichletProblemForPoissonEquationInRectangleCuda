#include "hip/hip_runtime.h"
#include <Std.h>
#include <CudaSupport.h>
#include <Definitions.h>
#include <CudaObjects.h>
#include <kernel.h>

///////////////////////////////////////////////////////////////////////////////

__device__ void BlockReduceMax( volatile NumericType* shared,
	const size_t threadIndex, const NumericType value )
{
	shared[threadIndex] = value;
	__syncthreads();
	if( threadIndex < 256 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 256] );
	}
	__syncthreads();
	if( threadIndex < 128 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 128] );
	}
	__syncthreads();
	if( threadIndex < 64 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 128] );
	}
	__syncthreads();
	if( threadIndex < 32 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 32] );
	}
	__syncthreads();
	if( threadIndex < 16 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 16] );
	}
	__syncthreads();
	if( threadIndex < 8 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 8] );
	}
	__syncthreads();
	if( threadIndex < 4 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 4] );
	}
	__syncthreads();
	if( threadIndex < 2 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 2] );
	}
	__syncthreads();
	if( threadIndex < 1 ) {
		shared[threadIndex] = max( shared[threadIndex], shared[threadIndex + 1] );
	}
}

__device__ void BlockReduceSumTwo( volatile NumericType* shared,
	const size_t threadIndex, const NumericType value1, const NumericType value2 )
{
	shared[threadIndex] = value1;
	shared[threadIndex + 1] = value2;
	__syncthreads();
	if( threadIndex < 512 ) {
		shared[threadIndex] += shared[threadIndex + 512];
		shared[threadIndex + 1] += shared[threadIndex + 512 + 1];
	}
	__syncthreads();
	if( threadIndex < 256 ) {
		shared[threadIndex] += shared[threadIndex + 256];
		shared[threadIndex + 1] += shared[threadIndex + 256 + 1];
	}
	__syncthreads();
	if( threadIndex < 128 ) {
		shared[threadIndex] += shared[threadIndex + 128];
		shared[threadIndex + 1] += shared[threadIndex + 128 + 1];
	}
	__syncthreads();
	if( threadIndex < 64 ) {
		shared[threadIndex] += shared[threadIndex + 64];
		shared[threadIndex + 1] += shared[threadIndex + 64 + 1];
	}
	__syncthreads();
	if( threadIndex < 32 ) {
		shared[threadIndex] += shared[threadIndex + 32];
		shared[threadIndex + 1] += shared[threadIndex + 32 + 1];
	}
	__syncthreads();
	if( threadIndex < 16 ) {
		shared[threadIndex] += shared[threadIndex + 16];
		shared[threadIndex + 1] += shared[threadIndex + 16 + 1];
	}
	__syncthreads();
	if( threadIndex < 8 ) {
		shared[threadIndex] += shared[threadIndex + 8];
		shared[threadIndex + 1] += shared[threadIndex + 8 + 1];
	}
	__syncthreads();
	if( threadIndex < 4 ) {
		shared[threadIndex] += shared[threadIndex + 4];
		shared[threadIndex + 1] += shared[threadIndex + 4 + 1];
	}
	__syncthreads();
	if( threadIndex < 2 ) {
		shared[threadIndex] += shared[threadIndex + 2];
		shared[threadIndex + 1] += shared[threadIndex + 2 + 1];
	}
}

///////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel( cudaMatrix arr, cudaMatrix result )
{
	extern __shared__ NumericType shared[];
	const size_t index = blockDim.x * blockIdx.x + threadIdx.x;
	BlockReduceMax( shared, threadIdx.x, arr( index, 0 ) );
	if( threadIdx.x == 0 ) {
		result( blockIdx.x, 0 ) = shared[0];
	}
}

__global__ void ReduceSumTwoKernel( cudaMatrix arr2, cudaMatrix result )
{
	extern __shared__ NumericType shared[];
	const size_t index = blockDim.x * blockIdx.x + threadIdx.x;
	BlockReduceSumTwo( shared, threadIdx.x * 2, arr2( index, 0 ), arr2( index, 1 ) );
	if( threadIdx.x == 0 ) {
		result( blockIdx.x, 0 ) = shared[0];
		result( blockIdx.x, 1 ) = shared[1];
	}
}

///////////////////////////////////////////////////////////////////////////////

__device__ NumericType LaplasOperator( cudaMatrix matrix, cudaUniformGrid grid, size_t x, size_t y )
{
	const NumericType ldx = ( matrix( x, y ) - matrix( x - 1, y ) ) / grid.X.Step( x - 1 );
	const NumericType rdx = ( matrix( x + 1, y ) - matrix( x, y ) ) / grid.X.Step( x );
	const NumericType tdy = ( matrix( x, y ) - matrix( x, y - 1 ) ) / grid.Y.Step( y - 1 );
	const NumericType bdy = ( matrix( x, y + 1 ) - matrix( x, y ) ) / grid.Y.Step( y );
	const NumericType dx = ( ldx - rdx ) / grid.X.AverageStep( x );
	const NumericType dy = ( tdy - bdy ) / grid.Y.AverageStep( y );
	return ( dx + dy );
}

///////////////////////////////////////////////////////////////////////////////

// Вычисление невязки rij во внутренних точках.
__global__ void kernelCalcR( cudaMatrix p, cudaUniformGrid grid, cudaMatrix r )
{
	const size_t x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	const size_t y = blockDim.y * blockIdx.y + threadIdx.y + 1;
	
	if( x < ( p.SizeX() - 1 ) && y < ( p.SizeY() - 1 ) ) {
		r( x, y ) = LaplasOperator( p, grid, x, y ) - F( grid.X[x], grid.Y[y] );
	}
}

// Вычисление значений gij во внутренних точках.
__global__ void kernelCalcG( cudaMatrix r, const NumericType alpha, cudaMatrix g )
{
	const size_t x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	const size_t y = blockDim.y * blockIdx.y + threadIdx.y + 1;

	if( x < ( g.SizeX() - 1 ) && y < ( g.SizeY() - 1 ) ) {
		g( x, y ) = r( x, y ) - alpha * g( x, y );
	}
}

// Вычисление значений pij во внутренних точках, возвращается максимум норма.
__global__ void kernelCalcP( cudaMatrix g, const NumericType tau, cudaMatrix p,
	cudaMatrix differences )
{
	extern __shared__ NumericType shared[];

	const size_t x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	const size_t y = blockDim.y * blockIdx.y + threadIdx.y + 1;
	const size_t threadIndex = threadIdx.y * blockDim.x + threadIdx.x;

	NumericType difference = 0;
	if( x < ( p.SizeX() - 1 ) && y < ( p.SizeY() - 1 ) ) {
		const NumericType newValue = p( x, y ) - tau * g( x, y );
		difference = abs( newValue - p( x, y ) );
		p( x, y ) = newValue;
	}

	BlockReduceMax( shared, threadIndex, difference );

	if( threadIndex == 0 ) {
		const size_t blockIndex = gridDim.x * blockIdx.y + blockIdx.x;
		differences( blockIndex, 0 ) = shared[0];
	}
}

// Вычисление alpha.
__global__ void kernelCalcAlpha( cudaMatrix r, cudaMatrix g, cudaUniformGrid grid,
	cudaMatrix alphas )
{
	extern __shared__ NumericType shared[];

	const size_t x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	const size_t y = blockDim.y * blockIdx.y + threadIdx.y + 1;
	const size_t threadIndex = threadIdx.y * blockDim.x + threadIdx.x;

	NumericType numerator = 0;
	NumericType denominator = 0;
	if( x < ( r.SizeX() - 1 ) && y < ( r.SizeY() - 1 ) ) {
		const NumericType common = g( x, y ) * grid.X.AverageStep( x ) * grid.Y.AverageStep( y );
		numerator = LaplasOperator( r, grid, x, y ) * common;
		denominator = LaplasOperator( g, grid, x, y ) * common;
	}

	BlockReduceSumTwo( shared, threadIndex * 2, numerator, denominator );

	if( threadIndex == 0 ) {
		const size_t blockIndex = gridDim.x * blockIdx.y + blockIdx.x;
		alphas( blockIndex, 0 ) = shared[0];
		alphas( blockIndex, 1 ) = shared[1];
	}
}

// Вычисление tau.
__global__ void kernelCalcTau( cudaMatrix r, cudaMatrix g, cudaUniformGrid grid,
	cudaMatrix taus )
{
	extern __shared__ NumericType shared[];

	const size_t x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	const size_t y = blockDim.y * blockIdx.y + threadIdx.y + 1;
	const size_t threadIndex = threadIdx.y * blockDim.x + threadIdx.x;

	NumericType numerator = 0;
	NumericType denominator = 0;
	if( x < ( r.SizeX() - 1 ) && y < ( r.SizeY() - 1 ) ) {
		const NumericType common = g( x, y ) * grid.X.AverageStep( x ) * grid.Y.AverageStep( y );
		numerator = r( x, y ) * common;
		denominator = LaplasOperator( g, grid, x, y ) * common;
	}

	BlockReduceSumTwo( shared, threadIndex * 2, numerator, denominator );

	if( threadIndex == 0 ) {
		const size_t blockIndex = gridDim.x * blockIdx.y + blockIdx.x;
		taus( blockIndex, 0 ) = shared[0];
		taus( blockIndex, 1 ) = shared[1];
	}
}

///////////////////////////////////////////////////////////////////////////////

const size_t SharedMemSize = BlockDim.x * BlockDim.y * sizeof( NumericType );
const size_t SharedMem2Size = SharedMemSize * 2;
const dim3 LinearBlockDim( 512 );
const size_t LinearSharedMemSize = LinearBlockDim.x * sizeof( NumericType );
const size_t LinearSharedMem2Size = LinearSharedMemSize * 2;

inline NumericType CalcMax( dim3 gridDim, cudaMatrix buffer1, cudaMatrix buffer2 )
{
	//const dim3 linearGridDim( buffer2.SizeX() );
	//ReduceMaxKernel<<<linearGridDim, LinearBlockDim, LinearSharedMemSize>>>( buffer1, buffer2 );
	buffer2 = buffer1;

	vector<NumericType> differences( buffer2.SizeX() );
	buffer2.GetPart( CMatrixPart( 0, buffer2.SizeX(), 0, 1 ), differences );

	NumericType difference = 0;
	for( size_t i = 0; i < buffer2.SizeX(); i++ ) {
		difference = max( difference, differences[i] );
	}

	return difference;
}

inline CFraction CalcFraction( dim3 gridDim, cudaMatrix buffer1, cudaMatrix buffer2 )
{
	//const dim3 linearGridDim( buffer2.SizeX() );
	//ReduceSumTwoKernel<<<linearGridDim, LinearBlockDim, LinearSharedMem2Size>>>( buffer1, buffer2 );
	buffer2 = buffer1;

	vector<NumericType> values( buffer2.SizeX() * 2 );
	buffer2.GetPart( CMatrixPart( 0, buffer2.SizeX(), 0, 2 ), values );

	NumericType numerator = 0;
	NumericType denominator = 0;
	for( size_t i = 0; i < buffer2.SizeX(); i++ ) {
		numerator += values[i];
		denominator += values[i + buffer2.SizeX()];
	}
	return CFraction( numerator, denominator );
}

// Вычисление невязки rij во внутренних точках.
void CalcR( dim3 gridDim, cudaMatrix p, cudaUniformGrid grid, cudaMatrix r )
{
	kernelCalcR<<<gridDim, BlockDim>>>( p, grid, r );
}

// Вычисление значений gij во внутренних точках.
void CalcG( dim3 gridDim, cudaMatrix r, const NumericType alpha, cudaMatrix g )
{
	kernelCalcG<<<gridDim, BlockDim>>>( r, alpha, g );
}

// Вычисление значений pij во внутренних точках, возвращается максимум норма.
NumericType CalcP( dim3 gridDim,
	cudaMatrix g, const NumericType tau, cudaMatrix p,
	cudaMatrix buffer1, cudaMatrix buffer2 )
{
	kernelCalcP<<<gridDim, BlockDim, SharedMemSize>>>( g, tau, p, buffer1 );
	return CalcMax( gridDim, buffer1, buffer2 );
}

// Вычисление alpha.
CFraction CalcAlpha( dim3 gridDim,
	cudaMatrix r, cudaMatrix g, cudaUniformGrid grid,
	cudaMatrix buffer1, cudaMatrix buffer2 )
{
	kernelCalcAlpha<<<gridDim, BlockDim, SharedMem2Size>>>( r, g, grid, buffer1 );
	return CalcFraction( gridDim, buffer1, buffer2 );
}

// Вычисление tau.
CFraction CalcTau( dim3 gridDim,
	cudaMatrix r, cudaMatrix g, cudaUniformGrid grid,
	cudaMatrix buffer1, cudaMatrix buffer2 )
{
	kernelCalcTau<<<gridDim, BlockDim, SharedMem2Size>>>( r, g, grid, buffer1 );
	return CalcFraction( gridDim, buffer1, buffer2 );
}

///////////////////////////////////////////////////////////////////////////////

#if 0

__global__ void CalcRuni( cudaMatrix p, cudaUniformGrid grid, cudaMatrix r )
{
	const size_t xPerThread = 0;
	const size_t yPerThread = 0;

	size_t x = ( BlockSizeX * blockIdx.x + threadIdx.x ) * xPerThread;
	const size_t xEnd = min( x + xPerThread, p.SizeX() );
	size_t y = ( BlockSizeY * blockIdx.y + threadIdx.y ) * yPerThread;
	const size_t yEnd = min( y + yPerThread, p.SizeY() );

	for( ; x < xEnd; x++ ) {
		for( ; y < yEnd; y++ ) {
			r( x, y ) = LaplasOperator( p, grid, x, y ) - F( grid.X[x], grid.Y[y] );
		}
	}
}

#endif
